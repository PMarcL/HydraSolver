#include "BinaryArithmeticIncrementalGPUFilter.cuh"
#include "BinaryArithmeticConstraint.h"
#include "Variable.h"
#include "BitsetIntVariable.h"
#include "hip/hip_runtime.h"
#include "BinaryArithmeticKernels.cuh"

using namespace std;

namespace hydra {

	BinaryArithmeticIncrementalGPUFilter::BinaryArithmeticIncrementalGPUFilter(BitsetIntVariable* var1, BitsetIntVariable* var2,
		Operator op, RelationalOperator relop, int rhs) : var1(var1), var2(var2), op(op), relop(relop), rhs(rhs), var1_lb(var1->getLowerBound()),
		var1_ub(var1->getUpperBound()), var2_lb(var2->getLowerBound()), var2_ub(var2->getUpperBound()) {
		hipMalloc((void**)&device_rhs, sizeof(int));
		hipMemcpy(device_rhs, &rhs, sizeof(int), hipMemcpyHostToDevice);

		auto temp = var1->getOriginalLowerBound();
		hipMalloc((void**)&deviceVar1Original_lb, sizeof(int));
		hipMemcpy(deviceVar1Original_lb, &temp, sizeof(int), hipMemcpyHostToDevice);

		temp = var2->getOriginalLowerBound();
		hipMalloc((void**)&deviceVar2Original_lb, sizeof(int));
		hipMemcpy(deviceVar2Original_lb, &temp, sizeof(int), hipMemcpyHostToDevice);

		hipMalloc((void**)&deviceVar1_lb, sizeof(int));
		hipMemcpy(deviceVar1_lb, &var1_lb, sizeof(int), hipMemcpyHostToDevice);
		
		hipMalloc((void**)&deviceVar2_lb, sizeof(int));
		hipMemcpy(deviceVar2_lb, &var2_lb, sizeof(int), hipMemcpyHostToDevice);

		hipMalloc((void**)&deviceVar1_ub, sizeof(int));
		hipMemcpy(deviceVar1_ub, &var1_ub, sizeof(int), hipMemcpyHostToDevice);
		
		hipMalloc((void**)&deviceVar2_ub, sizeof(int));
		hipMemcpy(deviceVar2_ub, &var2_ub, sizeof(int), hipMemcpyHostToDevice);

		unsigned int size = var1->getOriginalSize();
		hipMalloc((void**)&deviceVar1_size, sizeof(unsigned int));
		hipMemcpy(deviceVar1_size, &size, sizeof(unsigned int), hipMemcpyHostToDevice);

		size = var2->getOriginalSize();
		hipMalloc((void**)&deviceVar2_size, sizeof(unsigned int));
		hipMemcpy(deviceVar2_size, &size, sizeof(unsigned int), hipMemcpyHostToDevice);

		auto sizeVar1 = var1->getOriginalSize();
		bitset_host_var1 = (uint8_t *)malloc(sizeVar1);
		hipMalloc((void **)&bitset_device_var1, sizeVar1);

		auto sizeVar2 = var2->getOriginalSize();
		bitset_host_var2 = (uint8_t *)malloc(sizeVar2);
		hipMalloc((void **)&bitset_device_var2, sizeVar2);

		hipMalloc((void**)&bitset_matrix_var1, sizeVar1 * sizeVar2 * sizeof(uint8_t));
		hipMalloc((void**)&bitset_matrix_var2, sizeVar2 * sizeVar1 * sizeof(uint8_t));
	}

	BinaryArithmeticIncrementalGPUFilter::~BinaryArithmeticIncrementalGPUFilter() {
		hipFree(deviceVar1_lb);
		hipFree(deviceVar2_lb);
		hipFree(deviceVar1_ub);
		hipFree(deviceVar2_ub);
		hipFree(device_rhs);
		hipFree(deviceVar1Original_lb);
		hipFree(deviceVar2Original_lb);
		hipFree(bitset_device_var1);
		hipFree(bitset_device_var2);
		free(bitset_host_var1);
		free(bitset_host_var2);
	}

	vector<Variable*> BinaryArithmeticIncrementalGPUFilter::filterBoundsGPU() {
		updateVar2DeviceAttributes();
		vector<Variable*> filteredVariables;

		auto valueIsFirst_host = true;
		hipMemcpy(&valueIsFirst, &valueIsFirst_host, sizeof(bool), hipMemcpyHostToDevice);
		if (filterVariableBounds(var1, deviceVar2_lb, deviceVar2_ub, deviceVar1Original_lb, bitset_device_var1, bitset_host_var1)) {
			filteredVariables.push_back(var1);
		}

		if (var1->cardinality() == 0) {
			return filteredVariables;
		}

		updateVar1DeviceAttributes();
		valueIsFirst_host = false;
		hipMemcpy(&valueIsFirst, &valueIsFirst_host, sizeof(bool), hipMemcpyHostToDevice);
		if (filterVariableBounds(var2, deviceVar1_lb, deviceVar1_ub, deviceVar2Original_lb, bitset_device_var2, bitset_host_var2)) {
			filteredVariables.push_back(var2);
		}

		return filteredVariables;
	}

	void BinaryArithmeticIncrementalGPUFilter::updateVar1DeviceAttributes() {
		if (var1_lb != var1->getLowerBound()) {
			var1_lb = var1->getLowerBound();
			hipMemcpy(deviceVar1_lb, &var1_lb, sizeof(int), hipMemcpyHostToDevice);
		}
		if (var1_ub != var1->getUpperBound()) {
			var1_ub = var1->getUpperBound();
			hipMemcpy(deviceVar1_ub, &var1_ub, sizeof(int), hipMemcpyHostToDevice);
		}
	}
	
	void BinaryArithmeticIncrementalGPUFilter::updateVar2DeviceAttributes() {
		if (var2_lb != var2->getLowerBound()) {
			var2_lb = var2->getLowerBound();
			hipMemcpy(deviceVar2_lb, &var2_lb, sizeof(int), hipMemcpyHostToDevice);
		}
		if (var2_ub != var2->getUpperBound()) {
			var2_ub = var2->getUpperBound();
			hipMemcpy(deviceVar2_ub, &var2_ub, sizeof(int), hipMemcpyHostToDevice);
		}
	}

	bool BinaryArithmeticIncrementalGPUFilter::filterVariableBounds(BitsetIntVariable* var, int *lb, int *ub, int *originalLowerBound, uint8_t *bitset_device,
		uint8_t *bitset_host) const {
		unsigned int size = var->getOriginalSize();
		switch (op) {
		case PLUS:
			switch (relop) {
			case EQ:
				filterBoundPLUS_EQ << <1, size >> > (device_rhs, lb, ub, originalLowerBound, bitset_device);
				break;
			case NEQ:
				filterBoundPLUS_NEQ << <1, size >> > (device_rhs, lb, ub, originalLowerBound, bitset_device);
				break;
			case GEQ:
				filterBoundPLUS_GEQ << <1, size >> > (device_rhs, lb, ub, originalLowerBound, bitset_device);
				break;
			case GT:
				filterBoundPLUS_GT << <1, size >> > (device_rhs, lb, ub, originalLowerBound, bitset_device);
				break;
			case LEQ:
				filterBoundPLUS_LEQ << <1, size >> > (device_rhs, lb, ub, originalLowerBound, bitset_device);
				break;
			case LT:
				filterBoundPLUS_LT << <1, size >> > (device_rhs, lb, ub, originalLowerBound, bitset_device);
				break;
			}
			break;
		case MINUS:
			switch (relop) {
			case EQ:
				filterBoundMINUS_EQ << <1, size >> > (device_rhs, lb, ub, originalLowerBound, bitset_device);
				break;
			case NEQ:
				filterBoundMINUS_NEQ << <1, size >> > (device_rhs, lb, ub, originalLowerBound, bitset_device);
				break;
			case GEQ:
				filterBoundMINUS_GEQ << <1, size >> > (device_rhs, lb, ub, originalLowerBound, bitset_device);
				break;
			case GT:
				filterBoundMINUS_GT << <1, size >> > (device_rhs, lb, ub, originalLowerBound, bitset_device);
				break;
			case LEQ:
				filterBoundMINUS_LEQ << <1, size >> > (device_rhs, lb, ub, originalLowerBound, bitset_device);
				break;
			case LT:
				filterBoundMINUS_LT << <1, size >> > (device_rhs, lb, ub, originalLowerBound, bitset_device);
				break;
			}
			break;
		case MULTIPLIES:
			switch (relop) {
			case EQ:
				filterBoundMULTIPLIES_EQ << <1, size >> > (device_rhs, lb, ub, originalLowerBound, bitset_device);
				break;
			case NEQ:
				filterBoundMULTIPLIES_NEQ << <1, size >> > (device_rhs, lb, ub, originalLowerBound, bitset_device);
				break;
			case GEQ:
				filterBoundMULTIPLIES_GEQ << <1, size >> > (device_rhs, lb, ub, originalLowerBound, bitset_device);
				break;
			case GT:
				filterBoundMULTIPLIES_GT << <1, size >> > (device_rhs, lb, ub, originalLowerBound, bitset_device);
				break;
			case LEQ:
				filterBoundMULTIPLIES_LEQ << <1, size >> > (device_rhs, lb, ub, originalLowerBound, bitset_device);
				break;
			case LT:
				filterBoundMULTIPLIES_LT << <1, size >> > (device_rhs, lb, ub, originalLowerBound, bitset_device);
				break;
			}
			break;
		case DIVIDES:
			switch (relop) {
			case EQ:
				filterBoundDIVIDES_EQ << <1, size >> > (device_rhs, lb, ub, originalLowerBound, bitset_device);
				break;
			case NEQ:
				filterBoundDIVIDES_NEQ << <1, size >> > (device_rhs, lb, ub, originalLowerBound, bitset_device);
				break;
			case GEQ:
				filterBoundDIVIDES_GEQ << <1, size >> > (device_rhs, lb, ub, originalLowerBound, bitset_device);
				break;
			case GT:
				filterBoundDIVIDES_GT << <1, size >> > (device_rhs, lb, ub, originalLowerBound, bitset_device);
				break;
			case LEQ:
				filterBoundDIVIDES_LEQ << <1, size >> > (device_rhs, lb, ub, originalLowerBound, bitset_device);
				break;
			case LT:
				filterBoundDIVIDES_LT << <1, size >> > (device_rhs, lb, ub, originalLowerBound, bitset_device);
				break;
			}
			break;
		}

		hipMemcpy(bitset_host, bitset_device, size, hipMemcpyDeviceToHost);

		return var->mergeBitset(bitset_host);
	}

	vector<Variable*> BinaryArithmeticIncrementalGPUFilter::filterDomainGPU() const {
		vector<Variable*> filteredVariables;

		auto valueIsFirst = true;
		bool *var1IsFirst;
		hipMalloc((void**)&var1IsFirst, sizeof(bool));
		hipMemcpy(var1IsFirst, &valueIsFirst, sizeof(bool), hipMemcpyHostToDevice);
		hipMemcpy(bitset_device_var2, var2->getBitset().data(), var2->getOriginalSize() * sizeof(uint8_t), hipMemcpyHostToDevice);
		if (filterVariableDomain(var1, var2, deviceVar1Original_lb, deviceVar2Original_lb, deviceVar2_size, bitset_device_var1,
			bitset_device_var2, bitset_host_var1, bitset_matrix_var1, var1IsFirst)) {
			filteredVariables.push_back(var1);
		}

		if (var1->cardinality() == 0) {
			return filteredVariables;
		}

		valueIsFirst = false;
		hipMemcpy(var1IsFirst, &valueIsFirst, sizeof(bool), hipMemcpyHostToDevice);
		hipMemcpy(bitset_device_var1, var1->getBitset().data(), var1->getOriginalSize() * sizeof(uint8_t), hipMemcpyHostToDevice);
		if (filterVariableDomain(var2, var1, deviceVar2Original_lb, deviceVar1Original_lb, deviceVar1_size, bitset_device_var2,
			bitset_device_var1, bitset_host_var2, bitset_matrix_var2, var1IsFirst)) {
			filteredVariables.push_back(var2);
		}
		return filteredVariables;
	}

	bool BinaryArithmeticIncrementalGPUFilter::filterVariableDomain(BitsetIntVariable* var1, BitsetIntVariable* var2, int *var1OriginalLowerBound, int *var2OriginalLowerBound,
		unsigned int *var2Size, uint8_t *bitsetDeviceVar1, uint8_t *bitsetDeviceVar2, uint8_t *bitsetHostVar1, uint8_t *bitset_matrix, bool *varIsFirst) const {
		unsigned int sizeVar1 = var1->getOriginalSize();
		unsigned int sizeVar2 = var2->getOriginalSize();
		dim3 dimBlock(sizeVar2, sizeVar1);

		filterDomainMINUS_NEQ << <1, dimBlock >> > (device_rhs, var2Size, var1OriginalLowerBound, var2OriginalLowerBound, bitsetDeviceVar2, bitset_matrix, varIsFirst);
		sumMatrixRows << <1, sizeVar1 >> > (bitset_matrix, var2Size, bitsetDeviceVar1);

		hipMemcpy(bitsetHostVar1, bitsetDeviceVar1, sizeVar1, hipMemcpyDeviceToHost);
		return var1->mergeBitset(bitsetHostVar1);
	}

}
