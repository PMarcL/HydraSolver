#include "SumConstraintUtils.cuh"
#include "SumConstraintKernels.cuh"
#include "hip/hip_runtime.h"

void launchFilteringKernels(int nKernel, int sum, int lowerBoundSum, int upperBoundSum, int originalLowerBound, std::vector<uint8_t>* bitSetPtr) {
	uint8_t* deviceBitSetPtr;
	hipMalloc((void**)&deviceBitSetPtr, bitSetPtr->size() * sizeof(uint8_t));

	filterVariableKernel << < 1, nKernel >> > (sum, lowerBoundSum, upperBoundSum, originalLowerBound, deviceBitSetPtr);

	hipMemcpy(bitSetPtr->data(), deviceBitSetPtr, bitSetPtr->size() * sizeof(uint8_t), hipMemcpyDeviceToHost);
}
