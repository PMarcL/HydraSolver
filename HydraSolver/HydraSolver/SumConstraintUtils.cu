#include "SumConstraintUtils.cuh"
#include "SumConstraintKernels.cuh"
#include "SumConstraint.h"
#include "hip/hip_runtime.h"
#include ""

void launchFilteringKernels(
	int nKernel,
	int sum,
	int lowerBoundSum,
	int upperBoundSum,
	int originalLowerBound,
	std::vector<bool>* bitSetPtr) {

	bool* deviceBitSetPtr;
	hipMalloc((void**)&deviceBitSetPtr, bitSetPtr->size() * sizeof(bool));
	hipMemcpy(deviceBitSetPtr, bitSetPtr, bitSetPtr->size() * sizeof(bool), hipMemcpyHostToDevice);

	filterVariableKernel << < 1, nKernel >> > (
		sum,
		lowerBoundSum,
		upperBoundSum,
		originalLowerBound,
		deviceBitSetPtr
		);

	hipMemcpy(deviceBitSetPtr, bitSetPtr, bitSetPtr->size() * sizeof(bool), hipMemcpyDeviceToHost);

}
