#include "SumConstraintUtils.cuh"
#include "SumConstraintKernels.cuh"
#include "SumConstraint.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

void launchFilteringKernels(
	int nKernel,
	int sum,
	int lowerBoundSum,
	int upperBoundSum,
	int originalLowerBound,
	std::vector<uint8_t>* bitSetPtr) {

	uint8_t* deviceBitSetPtr;
	hipMalloc((void**)&deviceBitSetPtr, bitSetPtr->size() * sizeof(uint8_t));
	hipMemcpy(deviceBitSetPtr, bitSetPtr, bitSetPtr->size(), hipMemcpyHostToDevice);

	std::cout << "sum: " << sum << " lowerBoundSum: " << lowerBoundSum << " upperBoundSum: " << upperBoundSum << std::endl;
	filterVariableKernel << < 1, nKernel >> > (
		sum,
		lowerBoundSum,
		upperBoundSum,
		originalLowerBound,
		deviceBitSetPtr
		);

	hipMemcpy(bitSetPtr, deviceBitSetPtr, bitSetPtr->size() * sizeof(uint8_t), hipMemcpyDeviceToHost);

}
