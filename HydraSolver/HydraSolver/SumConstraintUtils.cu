#include "SumConstraintUtils.cuh"
#include "SumConstraintKernels.cuh"
#include "SumConstraint.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

void launchFilteringKernels(
	int nKernel,
	int sum,
	int lowerBoundSum,
	int upperBoundSum,
	int originalLowerBound,
	std::vector<bool>* bitSetPtr) {

	uint8_t* deviceBitSetPtr;
	hipMalloc((void**)&deviceBitSetPtr, bitSetPtr->size());
	hipMemcpy(deviceBitSetPtr, bitSetPtr, bitSetPtr->size(), hipMemcpyHostToDevice);

	std::cout << "sum: " << sum << " lowerBoundSum: " << lowerBoundSum << " upperBoundSum: " << upperBoundSum << std::endl;
	filterVariableKernel << < 1, nKernel >> > (
		sum,
		lowerBoundSum,
		upperBoundSum,
		originalLowerBound,
		deviceBitSetPtr
		);

	hipMemcpy(bitSetPtr, deviceBitSetPtr, bitSetPtr->size(), hipMemcpyDeviceToHost);

}
