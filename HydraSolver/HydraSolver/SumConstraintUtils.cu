#include "SumConstraintUtils.cuh"
#include "SumConstraintKernels.cuh"
#include "SumConstraint.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

void launchFilteringKernels(
	int nKernel,
	int sum,
	int lowerBoundSum,
	int upperBoundSum,
	int originalLowerBound,
	std::vector<uint8_t>* bitSetPtr) {

	uint8_t* deviceBitSetPtr;
	int* deviceModifiedPtr;
	hipMalloc((void**)&deviceBitSetPtr, bitSetPtr->size() * sizeof(uint8_t));
	hipMemcpy(deviceBitSetPtr, bitSetPtr->data(), bitSetPtr->size() * sizeof(uint8_t), hipMemcpyHostToDevice);

	filterVariableKernel << < 1, nKernel >> > (
		sum,
		lowerBoundSum,
		upperBoundSum,
		originalLowerBound,
		deviceBitSetPtr
		);

	hipMemcpy(bitSetPtr->data(), deviceBitSetPtr, bitSetPtr->size() * sizeof(uint8_t), hipMemcpyDeviceToHost);
}
