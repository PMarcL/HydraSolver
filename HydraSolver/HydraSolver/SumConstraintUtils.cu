#include "SumConstraintUtils.cuh"
#include "SumConstraintKernels.cuh"
#include "hip/hip_runtime.h"
#include "HydraException.h"

void launchFilteringKernels(int nKernel, int sum, int lowerBoundSum, int upperBoundSum, int originalLowerBound, std::vector<uint8_t>* bitSetPtr) {
	uint8_t* deviceBitSetPtr = nullptr;
	while (nKernel % 1024 != 0) {
		nKernel++;
	}

	auto error = hipMalloc((void**)&deviceBitSetPtr, nKernel * sizeof(uint8_t));
	if (error != hipSuccess) {
		throw hydra::HydraException("Cuda memory allocation error.");
	}
	auto nBlocks = nKernel / 1024;
	if (nBlocks > 65535) {
		throw hydra::HydraException("The domain is too big to filter on the GPU.");
	}
	filterVariableKernel << < nBlocks, 1024 >> > (sum, lowerBoundSum, upperBoundSum, originalLowerBound, deviceBitSetPtr);
	error = hipMemcpy(bitSetPtr->data(), deviceBitSetPtr, bitSetPtr->size() * sizeof(uint8_t), hipMemcpyDeviceToHost);
	if (error != hipSuccess) {
		throw hydra::HydraException("Cuda memory copy error");
	}
	error = hipFree(deviceBitSetPtr);
	if (error != hipSuccess) {
		throw hydra::HydraException("Cuda free memory error");
	}
}
