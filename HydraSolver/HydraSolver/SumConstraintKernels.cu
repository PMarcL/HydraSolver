#include "hip/hip_runtime.h"
﻿#include "SumConstraintKernels.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <vector>

__global__ void filterVariableKernel(
	int sum,
	int lowerBoundSum,
	int upperBoundSum,
	int offset,
	uint8_t* pBitset
) {
	int value = threadIdx.x + offset;
	lowerBoundSum += value;
	upperBoundSum += value;
	auto hasSupport = !(sum < lowerBoundSum || sum > upperBoundSum);
	pBitset[threadIdx.x] = uint8_t(bool(pBitset[threadIdx.x]) && hasSupport);
	return;
}