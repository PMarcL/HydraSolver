#include "hip/hip_runtime.h"
﻿#include "SumConstraintKernels.cuh"
#include "hip/hip_runtime.h"
#include ""

__global__ void filterVariableKernel(int sum, int lowerBoundSum, int upperBoundSum, int originalLowerBound, uint8_t* pBitset) {
	int value = threadIdx.x + 1024 * blockIdx.x + originalLowerBound;
	lowerBoundSum += value;
	upperBoundSum += value;
	auto hasSupport = sum >= lowerBoundSum && sum <= upperBoundSum;
	pBitset[threadIdx.x + 1024 * blockIdx.x] = uint8_t(hasSupport);
}