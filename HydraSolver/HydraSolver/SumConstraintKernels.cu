#include "hip/hip_runtime.h"
﻿#include "SumConstraintKernels.cuh"
#include "hip/hip_runtime.h"
#include ""

__global__ void filterVariableKernel(int sum, int lowerBoundSum, int upperBoundSum, int originalLowerBound, uint8_t* pBitset) {
	int value = threadIdx.x + 1024 * threadIdx.y + originalLowerBound;
	lowerBoundSum += value;
	upperBoundSum += value;
	auto hasSupport = sum >= lowerBoundSum && sum <= upperBoundSum;
	pBitset[1024 * threadIdx.y + threadIdx.x] = uint8_t(hasSupport);
}